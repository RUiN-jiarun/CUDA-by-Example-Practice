#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "book.h"

#define N 10

__global__ void add(int* a, int* b, int* c)
{
	int tid = blockIdx.x;	// �����������������
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}


int main()
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;

	// ��GPU�Ϸ����ڴ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// ��CPU��Ϊ����a[] b[]��ֵ
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	
	// a b���Ƶ�GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	// ����������ִ��add()�е��豸����
	// �����˲����߳̿������ΪN
	add << <N, 1 >> > (dev_a, dev_b, dev_c);

	// c���Ƶ�CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// �ͷ�GPU�ڴ�
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}