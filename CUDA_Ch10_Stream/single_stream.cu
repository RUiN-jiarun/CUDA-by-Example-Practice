#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "book.h"

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)


/// <summary>
/// �˺���������a������ֵ��b������ֵ��ƽ��ֵ
/// </summary>
/// <param name="a"></param>
/// <param name="b"></param>
/// <param name="c"></param>
/// <returns></returns>
__global__ void kernel(int* a, int* b, int* c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void)
{
    hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    // ѡ��֧���豸�ص����ܵ��豸������ִ��һ��CUDA�˺�����ͬʱ���������豸������֮��ִ�и��Ʋ���
    if (!prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream;
    int* host_a, * host_b, * host_c;
    int* dev_a, * dev_b, * dev_c;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // ��ʼ����
    HANDLE_ERROR(hipStreamCreate(&stream));

    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // ��������ʹ�õ�ҳ�����ڴ�
    // ��ʹ��hipHostAlloc()���������ϵĹ̶��ڴ�
    HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR(hipEventRecord(start, 0));
    // �����뻺��������Ϊ��С�Ŀ飬����ÿ������ִ��
    // ������������ѭ����ÿ�����ݿ��СΪN
    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
        // �������ڴ����첽��ʽ���Ƶ��豸��
        // hipMemcpy()��ͬ����ʽִ�У�����������ʱ�����Ʋ����Ѿ���ɣ�����������������а����˸��ƽ�ȥ������
        // hipMemcpyAsync()ֻ��һ������ͨ������streamָ��������������ʱֻ��ȷ����ֵ�����ᱻ����һ���������еĲ���֮ǰִ��
        // ֻ�����첽��ʽ��ҳ�����ڴ���и��Ʋ���
        HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

        kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);

        // �����ݴ��豸���Ƶ������ڴ�
        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));

    }

    // ����������ҳ�����ڴ渴�Ƶ������ڴ�
    HANDLE_ERROR(hipStreamSynchronize(stream));

    HANDLE_ERROR(hipEventRecord(stop, 0));

    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
        start, stop));
    printf("Time taken:  %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    HANDLE_ERROR(hipStreamDestroy(stream));

    return 0;
}

// 42.8ms