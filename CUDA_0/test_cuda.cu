#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void hello_world_from_gpu(void)
{
	printf("Hello World from GPU\n");
	return;
}

void test()
{
	printf("Hello World from CPU\n");
	hello_world_from_gpu << < 1, 5 >> > ();
	hipDeviceReset();
}

int main()
{
	test();
	return 0;
}