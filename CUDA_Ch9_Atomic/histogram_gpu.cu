#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "device_atomic_functions.hpp"
#include <stdio.h>
#include <stdlib.h>
//#include "hip/hip_runtime.h"
#include "book.h"

#define SIZE    (100*1024*1024)

/// <summary>
/// ʹ��ȫ���ڴ�ԭ�Ӳ���
/// </summary>
/// <param name="buffer"></param>
/// <param name="size"></param>
/// <param name="histo"></param>
/// <returns></returns>
//__global__ void hist_kernel(unsigned char* buffer, long size, unsigned int* hist)
//{
//    int i = threadIdx.x + blockIdx.x * blockDim.x;
//    int stride = blockDim.x * gridDim.x;
//    // ÿ���߳�֪����ʼƫ��i�͵����������������������飬����ֱ��ͼ����ӦԪ�ص�ֵ
//    while (i < size)
//    {
//        atomicAdd(&hist[buffer[i]], 1);         // CUDA��ԭ�Ӳ������Ը�λ�õ�ֵ����1
//        i += stride;
//    }
//}

/// <summary>
/// ʹ�ù����ڴ�ԭ�Ӳ�����ȫ���ڴ�ԭ�Ӳ���
/// </summary>
/// <param name="buffer"></param>
/// <param name="size"></param>
/// <param name="histo"></param>
/// <returns></returns>
__global__ void hist_kernel(unsigned char* buffer, long size, unsigned int* hist)
{

    // ���乲���ڴ滻��������ʼ��
    __shared__  unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();                        // ͬ����ȷ���ύ����д�����

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size)
    {
        atomicAdd(&temp[buffer[i]], 1);     
        i += stride;
    }
    // ��ÿ���߳̿����ʱֱ��ͼ�ϲ���ȫ�ֻ���hist[]
    __syncthreads();
    atomicAdd(&(hist[threadIdx.x]), temp[threadIdx.x]);     // ���߳̿��ֱ��ͼ��ÿ��Ԫ�ض��ӵ�����ֱ��ͼ����Ӧλ�õ�Ԫ����
}


int main()
{
    unsigned char* buffer = (unsigned char*)big_random_block(SIZE);

    hipEvent_t  start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    unsigned char* dev_buffer;
    unsigned int* dev_hist;
    HANDLE_ERROR(hipMalloc((void**)&dev_buffer, SIZE));
    HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_hist, 256 * sizeof(int)));
    HANDLE_ERROR(hipMemset(dev_hist, 0, 256 * sizeof(int)));

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t  prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;
    hist_kernel << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_hist);

    unsigned int hist[256];
    HANDLE_ERROR(hipMemcpy(hist, dev_hist, 256 * sizeof(int), hipMemcpyDeviceToHost));

    // get stop time, and display the timing results
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);

    long histCount = 0;
    for (int i = 0; i < 256; i++)
    {
        histCount += hist[i];
    }
    printf("Histogram Sum:  %ld\n", histCount);

    // ��֤��CPU������һ��
    for (int i = 0; i < SIZE; i++)
        hist[buffer[i]]--;
    for (int i = 0; i < 256; i++)
    {
        if (hist[i] != 0)
            printf("Failure at %d!  Off by %d\n", i, hist[i]);
    }

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    hipFree(dev_hist);
    hipFree(dev_buffer);
    free(buffer);
    return 0;
}